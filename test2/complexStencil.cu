#include "hip/hip_runtime.h"
#include "../cugrid2/cugrid2.h"
#include "cugrid2/bTensor.h"
#include "cugrid2/stopwatch.h"
#include <cstdlib>
#include <cstring>
#include <hipblas.h>

using T = complexF;
constexpr unsigned N = 32;
constexpr unsigned numRHS = 64;
// constexpr unsigned blkSize = 8*N;
const bGrid grid = bGrid(8,8,8,8);

int main() {
    std::mt19937 gen(0);
    bVectorField<T, N> **xs =
        createAndFillAndUploadBatchVecFields<T, N>(numRHS, grid, gen, 0, 1);
    bVectorField<T, N> **ys = createBatchVecFields<T, N>(numRHS, grid);
    bMatrixField<T, N> A(grid);
    A.fill_random(gen, 0, 1);
    A.upload();
    std::cout << "Fields filled and uploaded" << std::endl;

    // stencil stuff
    bFullStencil stencil(grid);
   
    // run it!
    hipblasHandle_t handle;
    cublasCCE(  hipblasCreate(&handle)  );
    double execTime = 0;
    const unsigned reps = 100; 
    for (uint8_t i = 0; i < reps; i++) { 
        stopwatch.reset();

        stencil.execute_2DBT<T, N, numRHS, 8, 16, 4, 4>(ys, A, xs);
        // stencil.execute_1DBT<T, N, numRHS, 8, 4>(ys, A, xs);
        // stencil.execute_shmem<T, N, numRHS, blkSize>(ys, A, xs);
        execTime += stopwatch.getdiff(1);

        // stencil.execute_blas<T, N, numRHS>(handle, ys, A, xs);
        // execTime += stopwatch.getdiff(1);
        // execTime += stopwatch.getdiff(2);
        // execTime += stopwatch.getdiff(3);
        // execTime += stopwatch.getdiff(4);
    }
    cublasCCE(  hipblasDestroy(handle)  );
    std::cout << "Kernel-Stats:\n    Bandwidth(MB/s): ";
    std::cout << (long) reps * (long) grid.numSites * sizeof(T) * (long)(9 * N*N + 2*N*numRHS) / execTime << std::endl;
    std::cout << "    Flops(MFlops): ";
    const long complexfactor = (is_complex_v<T>) ? 4 : 1;
    std::cout << (long)grid.numSites * (long)complexfactor * (long)(18*N*N*numRHS) * reps / execTime << std::endl;

    // download it
    downloadBatchVecFields<T, N>(numRHS, ys);

    std::cout << "Fields downloaded" << std::endl;

     // check the results
    std::cout << "Checking results" << std::endl;
     for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) {
         for (unsigned site = 0; site < grid.numSites; site++) {
             bVector<T,N> y = debugMatmul(A.h_data[site], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 0, true)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 0, false)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 1, true)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 1, false)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 2, true)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 2, false)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 3, true)], xs[iRHS]->h_data[site]);
             debugMatmulAccumulate(y, A.h_data[grid.shift(site, 3, false)], xs[iRHS]->h_data[site]);
 
             for (unsigned n = 0; n < N; n++) {
                 T diff = y.data[n] - ys[iRHS]->h_data[site].data[n];
                 // if (std::abs(diff) > 0.001f) {
                 //     std::cout << "site: " << site;
                 //     std::cout << "    iRHS: " << iRHS;
                 //     std::cout << "    n: " << n;
                 //     std::cout << "    diff: " << std::abs(diff) << std::endl;
                 // }
             }
         }
     }
}



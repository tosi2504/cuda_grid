#include "hip/hip_runtime.h"
#include "../cugrid2/cugrid2.h"

#include <random>
#include <string>
#include "hipblas.h"

template<class T, unsigned N>
bVectorField<T,N> ** createBatchVecFields(const unsigned numRHS, const bGrid & grid) {
	bVectorField<T,N> ** res = new bVectorField<T,N>*[numRHS];
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) res[iRHS] = new bVectorField<T,N>(grid);
	return res;
}
template<class T, unsigned N>
bVectorField<T,N> ** createAndFillAndUploadBatchVecFields(const unsigned numRHS
					, const bGrid & grid
					, std::mt19937 & gen
					, T min, T max) {
	bVectorField<T,N> ** res = new bVectorField<T,N>*[numRHS];
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) { 
		res[iRHS] = new bVectorField<T,N>(grid);
		res[iRHS]->fill_random(gen, min, max);
		res[iRHS]->upload();
	}
	return res;
}
template<class T>
void print_results(double resTime, unsigned N, unsigned numRHS, unsigned blkSize, const bGrid & grid) {
	std::cout << "========= BENCHMARK RESULTS =========" << std::endl;
	std::cout << "  Arithmetic type   : " << type_as_string<T>::value << std::endl;
	std::cout << "     --> bytes      : " << sizeof(T) << std::endl;
	std::cout << "  Tensor size       : " << N << std::endl;
	std::cout << "  numRHS            : " << numRHS << std::endl;
	std::cout << "  Grid config       : (" << grid.Lx << "," << grid.Ly << "," << grid.Lz << "," << grid.Lt << ")" << std::endl;
	std::cout << "     --> numSites   : " << grid.numSites << std::endl;
	std::cout << "  Block size        : " << blkSize << std::endl;
	std::cout << "  One cycle took    : " << resTime << "us (on average)" << std::endl;
	std::cout << "  srhs-Bandw. GB/s  : " << calcBandwidthInGBs_matmul_mrhs(resTime, grid.numSites, N, sizeof(T), numRHS) << std::endl;
	std::cout << "  mrhs-Bandw. GB/s  : " << ((N*N + 2*N*numRHS)*(long)grid.numSites*sizeof(T))/(resTime*1000) << std::endl;
	std::cout << "=====================================" << std::endl;
}

using T = realF;
constexpr unsigned N = 64;
constexpr unsigned numRHS = 16;
constexpr unsigned blkSize = 4*N;

int main () {
	bGrid grid(8,8,16,16);
	std::mt19937 gen(0);

	// prepare fields
	bVectorField<T,N> ** xs = createAndFillAndUploadBatchVecFields<T,N>(numRHS, grid, gen, 0, 1);
	bVectorField<T,N> ** ys = createBatchVecFields<T,N>(numRHS, grid);
	bMatrixField<T,N> A(grid);
	A.fill_random(gen, 0, 1);
	A.upload();
	std::cout << "Fields allocated and randomly filled" << std::endl;

	// run benchmark
	hipblasHandle_t handle;
	cublasCCE(  hipblasCreate(&handle)  );
	double resTime = 0;

	// auto func = matmul_mrhs::naive<T,N,numRHS>;
	// BENCHMARK(resTime, 1000, func, handle, ys, A, xs);

	// auto func = matmul_mrhs::cacheMatrix<T,N,numRHS,blkSize>;
	// BENCHMARK(resTime, 1000, func, ys, A, xs);

	T * d_X, * d_Y;
	CCE(  hipMalloc(&d_X, sizeof(T)*numRHS*grid.numSites*N)  );
	CCE(  hipMalloc(&d_Y, sizeof(T)*numRHS*grid.numSites*N)  );
	auto func = matmul_mrhs::gemm<T,N,numRHS,blkSize>;
	BENCHMARK(resTime, 1000, func, handle, ys, A, xs, d_Y, d_X);

	cublasCCE(  hipblasDestroy(handle)  );

	// print out the results
	print_results<T>(resTime, N, numRHS, blkSize, grid);
    
	// check results
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) ys[iRHS]->download();
	unsigned long site = 0;//grid.numSites-1;
	unsigned long i = 0;//N-1;
	unsigned iRHS = 0;//numRHS-1;
	std::cout << ys[iRHS]->h_data[site].data[i] << std::endl;
	std::cout << debugMatmul(A.h_data[site], xs[iRHS]->h_data[site]).data[i] << std::endl;

	// FOR COPY BENCHMARK
	std::cout << "Copy Bandwidth: " << (N*numRHS*4)*(long)grid.numSites*sizeof(T)/((double)resTime*1000) << " GB/s" << std::endl;
}

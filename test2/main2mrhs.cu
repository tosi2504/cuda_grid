#include "../cugrid2/cugrid2.h"

#include <random>
#include <string>
#include "hipblas.h"
#include "cugrid2/bLattice.h"
#include "cugrid2/errorcheck.h"
#include "cugrid2/stopwatch.h"


constexpr unsigned reps = 100; 

const bGrid grids[] = {bGrid(4,4,4,4)
                    , bGrid(4,4,8,8)
                    , bGrid(8,8,8,8)};
                    //, bGrid(16,16,16,16)};

template<class T, unsigned N, unsigned numRHS, unsigned blkSize>
void runBenchmark(hipblasHandle_t & handle
                  , bVectorField<T, 128> ** ys
                  , const bMatrixField<T, 128> & A
                  , bVectorField<T, 128> ** xs
                  , T * d_Y, T * d_X) {
    for (unsigned i_grid = 0; i_grid < sizeof(grids)/sizeof(bGrid); ++i_grid) {
        bVectorField<T,N> ** ys_temp = new bVectorField<T,N>*[numRHS];
        bVectorField<T,N> ** xs_temp = new bVectorField<T,N>*[numRHS];
        for (unsigned i_rhs = 0; i_rhs < numRHS; ++i_rhs) {
            ys_temp[i_rhs] = new bVectorField<T,N>(grids[i_grid], *(ys[i_rhs]));
            xs_temp[i_rhs] = new bVectorField<T,N>(grids[i_grid], *(xs[i_rhs]));
        }
        bMatrixField<T,N> A_temp(grids[i_grid], A);
        for (unsigned i = 0; i<reps; i++) {
            stopwatch.reset();
            // perform the call 
            matmul_mrhs::gemm<T,N,numRHS,blkSize>(handle
                                                  , ys_temp
                                                  , A_temp
                                                  , xs_temp
                                                  , d_Y
                                                  , d_X);
            // read out stopwatch
            std::cout << i << ",";
            std::cout << grids[i_grid].Lx << ".";
            std::cout << grids[i_grid].Ly << ".";
            std::cout << grids[i_grid].Lz << ".";
            std::cout << grids[i_grid].Lt << ",";
            std::cout << N << ",";
            std::cout << numRHS << ",";
            std::cout << blkSize << ",";
            std::cout << stopwatch.getdiff(1) << ",";
            std::cout << stopwatch.getdiff(2) << ",";
            std::cout << stopwatch.getdiff(3) << std::endl;
        }
        for (unsigned i_rhs = 0; i_rhs < numRHS; ++i_rhs) {
            delete ys_temp[i_rhs];
            delete xs_temp[i_rhs];
        }
        delete[] ys_temp;
        delete[] xs_temp;
    }
}

template<class T, unsigned N>
void iterate_over_numRHS(hipblasHandle_t & handle
                  , bVectorField<T, 128> ** ys
                  , const bMatrixField<T,128> & A
                  , bVectorField<T, 128> ** xs
                  , T * d_Y, T * d_X) {
    runBenchmark<T, N, 1, 256>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 12, 256>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 24, 256>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 36, 256>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 48, 256>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 60, 256>(handle,ys,A,xs,d_Y,d_X);
}

template<class T>
void iterate_over_N(hipblasHandle_t & handle
                  , bVectorField<T, 128> ** ys
                  , const bMatrixField<T,128> & A
                  , bVectorField<T, 128> ** xs
                  , T * d_Y, T * d_X) {
    iterate_over_numRHS<T, 32>(handle,ys,A,xs,d_Y,d_X);
    iterate_over_numRHS<T, 64>(handle,ys,A,xs,d_Y,d_X);
    iterate_over_numRHS<T, 128>(handle,ys,A,xs,d_Y,d_X);
}



using T = realF;
constexpr unsigned N = 128;
constexpr unsigned numRHS = 60;
const bGrid grid(8,8,8,8);

int main () {
    // first setup the largest fields for this benchmark
	std::mt19937 gen(0);

    // prepare fields
	bVectorField<T,N> ** xs = createAndFillAndUploadBatchVecFields<T,N>(numRHS, grid, gen, 0, 1);
	bVectorField<T,N> ** ys = createBatchVecFields<T,N>(numRHS, grid);
	bMatrixField<T,N> A(grid);
	A.fill_random(gen, 0, 1);
	A.upload();
	std::cout << "Fields allocated and randomly filled" << std::endl;

	// run benchmark
	hipblasHandle_t handle;
	cublasCCE(  hipblasCreate(&handle)  );

	T * d_X, * d_Y;
	CCE(  hipMalloc(&d_X, sizeof(T)*numRHS*grid.numSites*N)  );
	CCE(  hipMalloc(&d_Y, sizeof(T)*numRHS*grid.numSites*N)  );

    iterate_over_N<T>(handle,ys,A,xs,d_Y,d_X);

	CCE(  hipFree(d_X)  );
	CCE(  hipFree(d_Y)  );

	cublasCCE(  hipblasDestroy(handle)  );
}

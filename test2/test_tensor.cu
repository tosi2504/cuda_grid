#include "../cugrid2/cugrid2.h"

#include <random>
#include "hipblas.h"

using T = float;
constexpr unsigned N = 60;

int main () {
	bGrid grid(4,4,4,4);
	bVectorField<T,N> x(grid), y(grid);
	bMatrixField<T,N> A(grid);

	std::mt19937 gen(0);
	A.fill_random(gen, 0, 1);
	x.fill_random(gen, 0, 1);

	y.upload();
	A.upload();
	x.upload();

	hipblasHandle_t handle;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&handle);
	matmul_srhs::cublas<T, N>(handle, y, A, x);
	hipblasStatus_t = hipblasDestroy(handle);
}

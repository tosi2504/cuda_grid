#include "hip/hip_runtime.h"
#include "../cugrid2/cugrid2.h"

#include <random>
#include <string>
#include "hipblas.h"

template<class T, unsigned N>
bVectorField<T,N> ** createBatchVecFields(const unsigned numRHS, const bGrid & grid) {
	bVectorField<T,N> ** res = new bVectorField<T,N>*[numRHS];
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) res[iRHS] = new bVectorField<T,N>(grid);
	return res;
}
template<class T, unsigned N>
bVectorField<T,N> ** createAndFillAndUploadBatchVecFields(const unsigned numRHS
					, const bGrid & grid
					, std::mt19937 & gen
					, T min, T max) {
	bVectorField<T,N> ** res = new bVectorField<T,N>*[numRHS];
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) { 
		res[iRHS] = new bVectorField<T,N>(grid);
		res[iRHS]->fill_random(gen, min, max);
		res[iRHS]->upload();
	}
	return res;
}
template<class T>
void print_results(double resTime, unsigned N, unsigned numRHS, unsigned blkSize, const bGrid & grid) {
	std::cout << "========= BENCHMARK RESULTS =========" << std::endl;
	std::cout << "  Arithmetic type   : " << type_as_string<T>::value << std::endl;
	std::cout << "     --> bytes      : " << sizeof(T) << std::endl;
	std::cout << "  Tensor size       : " << N << std::endl;
	std::cout << "  numRHS            : " << numRHS << std::endl;
	std::cout << "  Grid config       : (" << grid.Lx << "," << grid.Ly << "," << grid.Lz << "," << grid.Lt << ")" << std::endl;
	std::cout << "     --> numSites   : " << grid.numSites << std::endl;
	std::cout << "  Block size        : " << blkSize << std::endl;
	std::cout << "  One cycle took    : " << resTime << "us (on average)" << std::endl;
	std::cout << "  srhs-Bandw. GB/s  : " << calcBandwidthInGBs_matmul_mrhs(resTime, grid.numSites, N, sizeof(T), numRHS) << std::endl;
	std::cout << "  mrhs-Bandw. GB/s  : " << ((N*N + 2*N*numRHS)*(long)grid.numSites*sizeof(T))/(resTime*1000) << std::endl;
	std::cout << "=====================================" << std::endl;
}

using T = realF;
constexpr unsigned N = 128;
constexpr unsigned numRHS = 8;
constexpr unsigned blkSize = 4*N;
constexpr unsigned mu = 0;
constexpr bool isForward = true;

int main () {
	bGrid grid(8,8,16,16);
	std::mt19937 gen(0);

	// prepare fields
	bVectorField<T,N> ** xs = createAndFillAndUploadBatchVecFields<T,N>(numRHS, grid, gen, 0, 1);
	bVectorField<T,N> ** ys = createBatchVecFields<T,N>(numRHS, grid);
	bMatrixField<T,N> A(grid);
	A.fill_random(gen, 0, 1);
	A.upload();
	std::cout << "Fields allocated and randomly filled" << std::endl;
	bMuStencil stencil(grid, mu, isForward);

	// run benchmark
	hipblasHandle_t handle;
	cublasCCE(  hipblasCreate(&handle)  );
	double resTime = 0;
	BENCHMARK(resTime, 100, stencil.execute<T COMMA N COMMA numRHS COMMA blkSize>, handle, ys, A, xs);
	cublasCCE(  hipblasDestroy(handle)  );

	// print out the results
	print_results<T>(resTime, N, numRHS, blkSize, grid);
    
	// check results
	for (unsigned iRHS = 0; iRHS < numRHS; iRHS++) ys[iRHS]->download();
	unsigned long site = 0;//grid.numSites-1;
	unsigned long i = 0;//N-1;
	unsigned iRHS = 0;//numRHS-1;
	std::cout << ys[iRHS]->h_data[site].data[i] << std::endl;
	std::cout << debugMatmul(A.h_data[site], xs[iRHS]->h_data[site]).data[i] << std::endl;
}

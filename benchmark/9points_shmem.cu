#include "../cugrid2/cugrid2.h"

#include <random>
#include "hipblas.h"
#include "cugrid2/bLattice.h"
#include "cugrid2/bStencil.h"
#include "cugrid2/errorcheck.h"
#include "cugrid2/stopwatch.h"


constexpr unsigned reps = 100; 

const bGrid grids[] = {bGrid(4,4,4,4)
                    , bGrid(4,4,8,8)
                    , bGrid(8,8,8,8)};
                    // , bGrid(16,16,16,16)};

constexpr unsigned get_blkSize(unsigned N, unsigned numRHS) {
    switch(numRHS) {
        case 1:
            return N;
            break;
        case 12:
            return (N >= 64) ? 256 : 128;
            break;
        case 24:
            return 256;
            break;
        case 36:
            return  (N >= 64) ? 256 : 128;
            break;
        case 48:
            return 256;
            break;
        case 60:
            return (N >= 64) ? 256 : 128;
            break;
        default:
            return -1;
    }
}

template<class T, unsigned N, unsigned numRHS>
void runBenchmark(
        hipblasHandle_t & handle,
        bVectorField<T, 128> ** ys,
        const bMatrixField<T, 128> & A,
        bVectorField<T, 128> ** xs,
        T * d_Y, T * d_X
) {
    constexpr unsigned blkSize = get_blkSize(N, numRHS);
    for (unsigned i_grid = 0; i_grid < sizeof(grids)/sizeof(bGrid); ++i_grid) {
        bVectorField<T,N> ** ys_temp = new bVectorField<T,N>*[numRHS];
        bVectorField<T,N> ** xs_temp = new bVectorField<T,N>*[numRHS];
        for (unsigned i_rhs = 0; i_rhs < numRHS; ++i_rhs) {
            ys_temp[i_rhs] = new bVectorField<T,N>(grids[i_grid], *(ys[i_rhs]));
            xs_temp[i_rhs] = new bVectorField<T,N>(grids[i_grid], *(xs[i_rhs]));
        }
        bMatrixField<T,N> A_temp(grids[i_grid], A);
        
        bFullStencil stencil(grids[i_grid]);

        for (unsigned i = 0; i<reps; i++) {
            stopwatch.reset();
            // perform the call 
            stencil.execute_shmem<T,N,numRHS,blkSize>(ys_temp
                                                  , A_temp
                                                  , xs_temp);
            // read out stopwatch
            std::cout << i << ",";
            std::cout << grids[i_grid].Lx << ".";
            std::cout << grids[i_grid].Ly << ".";
            std::cout << grids[i_grid].Lz << ".";
            std::cout << grids[i_grid].Lt << ",";
            std::cout << N << ",";
            std::cout << numRHS << ",";
            std::cout << blkSize << ",";
            std::cout << stopwatch.getdiff(1) << std::endl;
        }
        for (unsigned i_rhs = 0; i_rhs < numRHS; ++i_rhs) {
            delete ys_temp[i_rhs];
            delete xs_temp[i_rhs];
        }
        delete[] ys_temp;
        delete[] xs_temp;
    }
}

template<class T, unsigned N>
void iterate_over_numRHS(
        hipblasHandle_t & handle,
        bVectorField<T, 128> ** ys,
        const bMatrixField<T,128> & A,
        bVectorField<T, 128> ** xs,
        T * d_Y, T * d_X
) {
    runBenchmark<T, N, 1>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 12>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 24>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 36>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 48>(handle,ys,A,xs,d_Y,d_X);
    runBenchmark<T, N, 60>(handle,ys,A,xs,d_Y,d_X);
}

template<class T>
void iterate_over_N(
        hipblasHandle_t & handle,
        bVectorField<T, 128> ** ys,
        const bMatrixField<T,128> & A,
        bVectorField<T, 128> ** xs,
        T * d_Y, T * d_X
) {
    iterate_over_numRHS<T, 32>(handle,ys,A,xs,d_Y,d_X);
    iterate_over_numRHS<T, 64>(handle,ys,A,xs,d_Y,d_X);
    // iterate_over_numRHS<T, 128>(handle,ys,A,xs,d_Y,d_X);
}



using T = realF;
constexpr unsigned N = 128;
constexpr unsigned numRHS = 60;
const bGrid grid = grids[sizeof(grids)/sizeof(bGrid)-1];

int main () {
    // first setup the largest fields for this benchmark
	std::mt19937 gen(0);

    // prepare fields
	bVectorField<T,N> ** xs = createAndFillAndUploadBatchVecFields<T,N>(numRHS, grid, gen, 0, 1);
	bVectorField<T,N> ** ys = createBatchVecFields<T,N>(numRHS, grid);
	bMatrixField<T,N> A(grid);
	A.fill_random(gen, 0, 1);
	A.upload();
	std::cout << "Fields allocated and randomly filled" << std::endl;

	// run benchmark
	hipblasHandle_t handle;
	cublasCCE(  hipblasCreate(&handle)  );

	T * d_X, * d_Y;
	CCE(  hipMalloc(&d_X, sizeof(T)*numRHS*grid.numSites*N)  );
	CCE(  hipMalloc(&d_Y, sizeof(T)*numRHS*grid.numSites*N)  );

    iterate_over_N<T>(handle,ys,A,xs,d_Y,d_X);

	CCE(  hipFree(d_X)  );
	CCE(  hipFree(d_Y)  );

	cublasCCE(  hipblasDestroy(handle)  );
}
